#include "hip/hip_runtime.h"
//
// Created by matteo on 11/4/21.
//

#include "HeatSimulation.cuh"

/* Translate a 2D index to a 1D index */
#define index(x, y) x * HEIGHT + y

/*
 * Construct a HeatSimulation
 */
HeatSimulation::HeatSimulation(int w, int h) : Engine(w, h) {
  host_grid = new Real[w * h];
  hipMalloc(&device_grid, w * h * sizeof(Real));
}

HeatSimulation::~HeatSimulation() {
  free(host_grid);
  hipFree(&device_grid);
}

/*
 * Gets called by the Engine at every render step
 * The screen is an SDL_Texture object
 * TODO: Refactor
 */
void HeatSimulation::render() {
  void *pixels;
  int pitch;
  Uint32 *dest;
  Real b;

  if (SDL_LockTexture(screen, nullptr, &pixels, &pitch) < 0) {
    SDL_LogError(SDL_LOG_CATEGORY_APPLICATION, "Couldn't lock texture: %s\n", SDL_GetError());
  }

  for (int y = 0; y < HEIGHT; y++) {
    dest = (Uint32 *) ((Uint8 *) pixels + y * pitch);
    for (int x = 0; x < WIDTH; x++) {
//      b = std::min(host_grid[index(x, y)], static_cast<Real>(1));
//      *(dest + x) = utils::HSBtoRGB(0.5, 1, b);
    }
  }
  SDL_UnlockTexture(screen);
}

/*
 * Gets called by the Engine at every simulation step
 * We need to calculate the temperature state at t + \delta t
 * Since I'm assuming that \delta x = \delta y = 1 to simplify things, then
 * to calculate T(x, y, t + \delta t ) = T(x, y - 1) + T(x - 1, y) - 4T(x, y) + T(x, y + 1) + T(x + 1, y)
 * Source:
 * http://www.u.arizona.edu/~erdmann/mse350/_downloads/2D_heat_equation.pdf
 */
void HeatSimulation::step() {

}

__global__ void step_kernel(Real *device_grid) {

}

