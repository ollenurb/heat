#include "hip/hip_runtime.h"
//
// Created by matteo on 11/19/21.
//
#include "Gpu.cuh"
#include <cstdio>
#define index(x, y) x * device::HEIGHT + y

dim3 dim_block;
dim3 dim_grid;

/*
 * Allocate just the HEIGHT of the simulation on the device
 */
namespace device {
  __device__ unsigned int HEIGHT;
  __device__ unsigned int WIDTH;
}

/*
 * Actually compute on the GPU
 */
__global__ void compute_kernel(Real *device_grid, Real gamma) {
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

  if((x > 0 && y > 0) && (x < device::WIDTH-1 && y < device::HEIGHT-1)) {
    unsigned int index = index(x, y);
    device_grid[index] = gamma *
                         (device_grid[index((x-1), y)] + device_grid[index((x+1), y)] +
                         (device_grid[index(x, (y-1))] + device_grid[index(x, (y+1))] -
                         (4 * device_grid[index(x, y)]))) + device_grid[index(x, y)];
  }
}

/*
 * Initialize the Device
 */
Gpu::Gpu(unsigned int w, unsigned int h, Real gamma, Real *host_grid) : HEIGHT(h), WIDTH(w), gamma(gamma) {
  hipMalloc(&device_grid, w * h * sizeof(Real));
  hipMemcpyToSymbol(HIP_SYMBOL(device::HEIGHT), &h, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL(device::WIDTH), &w, sizeof(unsigned int));
  hipMemcpy(device_grid, host_grid, sizeof(Real) * (WIDTH * HEIGHT), hipMemcpyHostToDevice);
  dim_block = dim3(10, 10);
  dim_grid = dim3(WIDTH / dim_block.x, HEIGHT / dim_block.y);
}

Gpu::~Gpu() {
  hipFree(&device_grid);
}

/*
 * Compute one step of simulation, then put the results into the host grid
 */
void Gpu::compute_step(Real *host_grid) {
  compute_kernel<<<dim_grid, dim_block>>>(device_grid, gamma);
  hipMemcpy(host_grid, device_grid, sizeof(Real) * (WIDTH * HEIGHT), hipMemcpyDeviceToHost);
}

void Gpu::mem_sync(Real *host_grid) {
  hipMemcpy(device_grid, host_grid, sizeof(Real) * (WIDTH * HEIGHT), hipMemcpyHostToDevice);
}
